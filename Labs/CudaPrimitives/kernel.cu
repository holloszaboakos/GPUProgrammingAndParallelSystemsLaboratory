﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <chrono>
#include <iostream>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__device__ void square(int& output, int input)
{
	output = input * input;
}

template <typename IN, typename OUT, void (FUN)(OUT&, IN)>
__global__ void map(IN* inputArray, OUT* outputArray)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	FUN(outputArray[index], inputArray[index]);
}

__device__ void sum(int& output, int left, int right)
{
	output = left + right;
}
template <typename T, void (FUN)(T&, T, T)>
__global__ void reduceFolding(T* inputArray, T* outputArray, unsigned int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	outputArray[index] = inputArray[index];
	__syncthreads();
	for (unsigned int shift = size / 2; shift != 0; shift /= 2) {
		if (index < shift)
			FUN(outputArray[index], outputArray[index], outputArray[index + shift]);
		__syncthreads();
	}
}
template <typename T, void (FUN)(T&, T, T)>
__global__ void reduceBinary(T* inputArray, T* outputArray, unsigned int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	outputArray[index] = inputArray[index];
	__syncthreads();
	for (unsigned int shift = 2; shift <= size; shift *= 2) {
		if (index % shift == 0)
			FUN(outputArray[index], outputArray[index], outputArray[index + (shift / 2)]);
		__syncthreads();
	}
}
template <typename T, void (FUN)(T&, T, T)>
__global__ void scanNaiv(T* inputArray, T* outputArray, unsigned int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	outputArray[index] = inputArray[index];
	__syncthreads();
	for (int shift = 1; shift <= size; shift *= 2) {
		if (index - shift >= 0)
			FUN(outputArray[index], outputArray[index], outputArray[index - shift]);
		__syncthreads();
	}
}

template <typename T, void (FUN)(T&, T, T)>
__global__ void scanBinary(T* inputArray, T* outputArray, unsigned int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	outputArray[index] = inputArray[index];
	__syncthreads();
	for (int shift = 2; shift <= size; shift *= 2) {
		int copySource = (shift / 2 - 1);
		if (index % shift > copySource)
			FUN(outputArray[index], outputArray[index], outputArray[index - (index % shift) + copySource]);
		__syncthreads();
	}
}

template <typename T>
__global__ void gather(T* inputArray, int* indexArray, T* outputArray)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	outputArray[index] = inputArray[indexArray[index]];
}

template <typename T>
__global__ void scatter(T* inputArray, int* indexArray, T* outputArray)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	outputArray[indexArray[index]] = inputArray[index];
}

template <typename T>
__global__ void compact(T* inputArray, int* flagArray, T* outputArray)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (flagArray[index] == 1) {
		int counter = 0;
		for (int i = 0; i < index; i++) {
			if (flagArray[i] == 1)
				counter++;
		}
		outputArray[counter] = inputArray[index];
	}
}

template <typename T>
__global__ void compactRewritingInputArray(T* dataArray, int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int value = dataArray[index];
	dataArray[index] = 0;
	if (dataArray[size + index] == 1) {
		int counter = 0;
		for (int i = size; i < size + index; i++) {
			if (dataArray[i] == 1)
				counter++;
		}
		dataArray[counter] = value;
	}
}

template <typename T>
__global__ void mergeSort(T* dataArray, T* swapArray, int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int value = dataArray[index];
	int firstArrayIndex;
	int secondArrayIndex;
	bool swap = true;
	for (int sortSize = 1; sortSize < size; sortSize *= 2) {
		if (index % (sortSize * 2) == 0) {
			firstArrayIndex = index;
			secondArrayIndex = index + sortSize;
			if (swap) {
				for (int resultIndex = index; resultIndex < index + sortSize * 2; resultIndex++)
					if (firstArrayIndex == index + sortSize) {
						swapArray[resultIndex] = dataArray[secondArrayIndex];
						secondArrayIndex++;
					}
					else if (secondArrayIndex == index + sortSize + sortSize) {
						swapArray[resultIndex] = dataArray[firstArrayIndex];
						firstArrayIndex++;
					}
					else if (dataArray[secondArrayIndex] < dataArray[firstArrayIndex]) {
						swapArray[resultIndex] = dataArray[secondArrayIndex];
						secondArrayIndex++;
					}
					else  {
						swapArray[resultIndex] = dataArray[firstArrayIndex];
						firstArrayIndex++;
					}
			}
			else {
				for (int resultIndex = index; resultIndex < index + sortSize * 2; resultIndex++)
					if (firstArrayIndex == index + sortSize) {
						dataArray[resultIndex] = swapArray[secondArrayIndex];
						secondArrayIndex++;
					}
					else if (secondArrayIndex == index + sortSize + sortSize) {
						dataArray[resultIndex] = swapArray[firstArrayIndex];
						firstArrayIndex++;
					}
					else if (swapArray[secondArrayIndex] < swapArray[firstArrayIndex]) {
						dataArray[resultIndex] = swapArray[secondArrayIndex];
						secondArrayIndex++;
					}
					else {
						dataArray[resultIndex] = swapArray[firstArrayIndex];
						firstArrayIndex++;
					}
			}
		}

		swap = !swap;
		__syncthreads();
	}
	if (!swap) {
		dataArray[index] = swapArray[index];
	}
}


__device__ int* data;

int main() {

	const int dataSize = 1024;
	int* dataCPU = new int[sizeof(int) * dataSize];
	int* indexCPU = new int[sizeof(int) * dataSize];
	int* resultCPU = new int[sizeof(int) * dataSize * 2];
	int* dataGPU;
	int* indexGPU;
	int* resultGPU;
	hipMalloc(&dataGPU, sizeof(int) * dataSize * 2);
	hipMalloc(&indexGPU, sizeof(int) * dataSize);
	hipMalloc(&resultGPU, sizeof(int) * dataSize * 2);
	int threadsPerBlock = 256;
	int blocksPerGrid = 4;
	int wrongCount = 0;
	
	
	//map test
	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = i;
	}
	hipMemcpy(dataGPU, dataCPU, sizeof(int) * dataSize, hipMemcpyHostToDevice);
	map<int, int, square> <<< blocksPerGrid, threadsPerBlock >>> (dataGPU, resultGPU);
	hipMemcpy(resultCPU, resultGPU, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
	for (int i = 0; i < dataSize; ++i) {
		if (resultCPU[i] != i * i) wrongCount++;
	}
	printf("Number of wrong squares : % d\n", wrongCount);
	//binary reduce test
	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = i;
	}
	hipMemcpy(dataGPU, dataCPU, sizeof(int) * dataSize, hipMemcpyHostToDevice);
	reduceBinary<int, sum> <<< blocksPerGrid, threadsPerBlock >>> (dataGPU, resultGPU, dataSize);
	hipMemcpy(resultCPU, resultGPU, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
	if (resultCPU[0] != (dataSize - 1 + 0) * dataSize / 2)
		printf("Wrong result : % d\n", resultCPU[0]);
	else
		printf("Good result\n");
	//folding reduce test
	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = i;
	}
	hipMemcpy(dataGPU, dataCPU, sizeof(int) * dataSize, hipMemcpyHostToDevice);
	reduceFolding<int, sum> <<< blocksPerGrid, threadsPerBlock >>> (dataGPU, resultGPU, dataSize);
	hipMemcpy(resultCPU, resultGPU, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
	if (resultCPU[0] != (dataSize - 1 + 0) * dataSize / 2)
		printf("Wrong result : % d\n", resultCPU[0]);
	else
		printf("Good result\n");

	//naiv scan test
	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = i;
	}
	hipMemcpy(dataGPU, dataCPU, sizeof(int) * dataSize, hipMemcpyHostToDevice);
	scanNaiv<int, sum> <<< blocksPerGrid, threadsPerBlock >>> (dataGPU, resultGPU, dataSize);
	hipMemcpy(resultCPU, resultGPU, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
	unsigned int sumOf = 0;
	wrongCount = 0;
	for (int i = 0; i < dataSize; ++i) {
		sumOf += i;
		if (resultCPU[i] != sumOf) {
			printf("result : % d\n", resultCPU[i]);
			printf("wanted : % d\n", sumOf);
			wrongCount++;
		}
	}
	printf("Number of wrong sums : % d\n", wrongCount);

	//binary scan test
	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = i;
	}
	hipMemcpy(dataGPU, dataCPU, sizeof(int) * dataSize, hipMemcpyHostToDevice);
	scanBinary<int, sum><<< blocksPerGrid, threadsPerBlock >>>(dataGPU, resultGPU, dataSize);
	hipMemcpy(resultCPU, resultGPU, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
	sumOf = 0;
	wrongCount = 0;
	for (int i = 0; i < dataSize; ++i) {
		sumOf += i;
		if (resultCPU[i] != sumOf) {
			printf("result : % d\n", resultCPU[i]);
			printf("wanted : % d\n", sumOf);
			wrongCount++;
		}
	}
	printf("Number of wrong sums : % d\n", wrongCount);

	//gather
	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = -i;
	}
	for (int i = 0; i < dataSize / 2; ++i) {
		indexCPU[i] = i * 2;
	}
	hipMemcpy(dataGPU, dataCPU, sizeof(int)* dataSize, hipMemcpyHostToDevice);
	hipMemcpy(indexGPU, indexCPU, sizeof(int) * (dataSize/2), hipMemcpyHostToDevice);
	gather<int> <<< blocksPerGrid / 2, threadsPerBlock >>> (dataGPU, indexGPU, resultGPU);
	hipMemcpy(resultCPU, resultGPU, sizeof(int) * (dataSize/2), hipMemcpyDeviceToHost);
	wrongCount = 0;
	for (int i = 0; i < dataSize / 2; ++i) {
		if (resultCPU[i] != -i * 2) {
			wrongCount++;
		}
	}
	printf("Number of wrong sums : % d\n", wrongCount);
	//scatter
	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = i;
	}
	for (int i = 0; i < dataSize; ++i) {
		indexCPU[i] = i * 2;
	}
	for (int i = 0; i < dataSize * 2; ++i) {
		resultCPU[i] = 0;
	}
	hipMemcpy(dataGPU, dataCPU, sizeof(int) * dataSize, hipMemcpyHostToDevice);
	hipMemcpy(indexGPU, indexCPU, sizeof(int)* dataSize, hipMemcpyHostToDevice);
	hipMemcpy(resultGPU, resultCPU, sizeof(int) * (dataSize * 2), hipMemcpyHostToDevice);
	scatter<int> <<< blocksPerGrid, threadsPerBlock >>> (dataGPU, indexGPU, resultGPU);
	hipMemcpy(resultCPU, resultGPU, sizeof(int) * (dataSize * 2), hipMemcpyDeviceToHost);
	wrongCount = 0;
	for (int i = 0; i < dataSize * 2; ++i) {
		if ((i%2 == 0 && resultCPU[i]!=i/2) || (i % 2 == 1 && resultCPU[i] != 0)) {
			wrongCount++;
		}
	}
	printf("Number of wrong sums : % d\n", wrongCount);
	
	//compact
	
	std::chrono::steady_clock::time_point initialize = std::chrono::steady_clock::now();
	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = i;
		indexCPU[i] = i % 2;
	}
	for (int i = 0; i < dataSize / 2; ++i) {
		resultCPU[i] = 0;
	}
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	hipMemcpy(dataGPU, dataCPU, sizeof(int) * dataSize, hipMemcpyHostToDevice);
	hipMemcpy(indexGPU, indexCPU, sizeof(int) * dataSize, hipMemcpyHostToDevice);
	compact<int> <<< blocksPerGrid, threadsPerBlock >>> (dataGPU, indexGPU, resultGPU);
	hipMemcpy(resultCPU, resultGPU, sizeof(int) * (dataSize / 2), hipMemcpyDeviceToHost);
	wrongCount = 0;
	for (int i = 0; i < dataSize / 2; ++i) {
		if (resultCPU[i] != (i * 2) + 1) {
			wrongCount++;
		}
	}
	printf("Number of wrong sums : % d\n", wrongCount);
	//gpu terület felszabadítása

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	std::cout << "Time until beging = " << std::chrono::duration_cast<std::chrono::nanoseconds> (begin - initialize).count() << "[nanoseconds]" << std::endl;
	std::cout << "Time until end = " << std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() << "[nanoseconds]" << std::endl;


	initialize = std::chrono::steady_clock::now();
	//const int dataSize = 1024;
	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = i;
	}
	for (int i = dataSize; i < dataSize * 2; ++i) {
		dataCPU[i] = i%2;
	}
	begin = std::chrono::steady_clock::now();
	hipMemcpy(dataGPU, dataCPU, sizeof(int) * dataSize * 2, hipMemcpyHostToDevice);
	compactRewritingInputArray<int> <<< blocksPerGrid, threadsPerBlock >>> (dataGPU, dataSize);
	hipMemcpy(dataCPU, dataGPU, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
	wrongCount = 0;
	for (int i = 0; i < dataSize / 2; ++i) {
		if (dataCPU[i] != (i * 2) + 1) {
			wrongCount++;
		}
	}
	for (int i = dataSize / 2; i < dataSize; ++i) {
		if (dataCPU[i] != 0) {
			wrongCount++;
		}
	}
	printf("Number of wrong sums : % d\n", wrongCount);
	//gpu terület felszabadítása
	//***C++11 Style:***

	end = std::chrono::steady_clock::now();

	std::cout << "Time until beging = " << std::chrono::duration_cast<std::chrono::nanoseconds> (begin - initialize).count() << "[nanoseconds]" << std::endl;
	std::cout << "Time until end = " << std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() << "[nanoseconds]" << std::endl;

	initialize = std::chrono::steady_clock::now();

	for (int i = 0; i < dataSize; ++i) {
		dataCPU[i] = dataSize - i;
	}
	begin = std::chrono::steady_clock::now();
	hipMemcpy(dataGPU, dataCPU, sizeof(int) * dataSize, hipMemcpyHostToDevice);
	mergeSort<int> <<< blocksPerGrid, threadsPerBlock >>> (dataGPU, dataGPU + dataSize, dataSize);
	hipMemcpy(dataCPU, dataGPU, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
	wrongCount = 0;
	for (int i = 0; i < dataSize - 1; ++i) {
		if (dataCPU[i] > dataCPU[i+1]) {
			wrongCount++;
			printf("index: % d value1: % d value2: % d\n", i, dataCPU[i], dataCPU[i+1]);
		}
	}
	printf("Number of wrong sums : % d\n", wrongCount);
	//gpu terület felszabadítása

	end = std::chrono::steady_clock::now();

	std::cout << "Time until beging = " << std::chrono::duration_cast<std::chrono::nanoseconds> (begin - initialize).count() << "[nanoseconds]" << std::endl;
	std::cout << "Time until end = " << std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() << "[nanoseconds]" << std::endl;

	hipFree(dataGPU);
	hipFree(indexGPU);
	hipFree(resultGPU);

	return 0;
}
