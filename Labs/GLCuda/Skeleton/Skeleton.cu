#include "hip/hip_runtime.h"
// Skeleton.cpp : Defines the entry point for the console application.
//

#include <GL/glew.h>
#include <GL/freeglut.h>
#include <glm/gtc/matrix_transform.hpp>

#include <cstdio>
#include <algorithm>

#include "shader.hpp"
#include "texture.hpp"
#include "DebugOpenGL.hpp"

#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>

const unsigned int windowWidth = 600;
const unsigned int windowHeight = 600;

// Number of particles
const unsigned int particlesNum = 256;

float posX;
float posY;

// Vec4 like structure
struct xyzw
{
	float x, y, z, w;
};

// Particle movement shader (compute shader)
Shader particleMoveShader;
// Particle renderer shaders (standard pipeline)
Shader particleRenderShader;

// Position buffer
GLuint positionBuffer;
// Velocity buffer
GLuint velocityBuffer;

// Vertex array object
GLuint vao;

hipGraphicsResource_t p_res = 0;
hipGraphicsResource_t v_res = 0;

__device__ float lengthVec(float* vector) {
	float sum;
	for (int index = 0; index < 3; index++)
		sum += vector[index] * vector[index];
	return sqrt(sum);
}

__device__ float* minusVec(float* output, float* leftVector, float* rightVector) {
	for (int index = 0; index < 3; index++)
		output[index] = leftVector[index] - rightVector[index];
	return output;
}

__device__ float* plusVec(float* output, float* leftVector, float* rightVector) {
	for (int index = 0; index < 3; index++)
		output[index] = leftVector[index] + rightVector[index];
	return output;
}

__device__ float* timesVec(float* output, float* leftVector, float rightValue) {
	for (int index = 0; index < 3; index++)
		output[index] = leftVector[index] * rightValue;
	return output;
}

__device__ float* assignVec(float* leftVector, float* rightVector) {
	for (int index = 0; index < 3; index++)
		leftVector[index] = rightVector[index];
	return leftVector;
}

__global__ void moveKernel(float* positionArray, float* velocityArray, int size) {

	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int startIndex = threadIndex * 4;

	const float dt = 0.0004;

	float newVelocity[3];
	assignVec(newVelocity, velocityArray + startIndex);
	float newPosition[3];
	float movement[3];
	plusVec(newPosition,
		positionArray + startIndex,
		timesVec(movement,
			newVelocity,
			dt
		)
	);


	bool shouldStepBack = false;
	for (int coordIndex = 0; coordIndex < 3; coordIndex++) {
		if (newPosition[coordIndex] <= -2.0 || newPosition[coordIndex] >= 2.0) {
			shouldStepBack = true;
			newVelocity[coordIndex] *= -1;
		}
	}

	if (shouldStepBack) {
		timesVec(newVelocity, newVelocity, 0.1);
		minusVec(newPosition, newPosition, movement);
	}


	for (int positionIndex = 0; positionIndex < size * 4; positionIndex += 4) {
		float acceleration[3];
		float epsilon = 0.01;

		minusVec(acceleration,
			positionArray + positionIndex,
			newPosition
		);

		float distance = lengthVec(acceleration);

		float temp = (distance * distance) + epsilon * epsilon;

		plusVec(newVelocity,
			newVelocity,
			timesVec(acceleration,
				acceleration,
				0.5 * dt / sqrt(temp * temp * temp)
			)
		);
	}

	__syncthreads();
	assignVec(positionArray + startIndex, newPosition);
	assignVec(velocityArray + startIndex, newVelocity);
}





void onInitialization()
{
	cudaGLSetGLDevice(0);
	glewExperimental = true;
	if (glewInit() != GLEW_OK)
	{
		printf("Cannot initialize GLEW\n");
		exit(-1);
	}
	glGetError();

	DebugOpenGL::init();
	DebugOpenGL::enableLowSeverityMessages(false);

	glClearColor(0.1f, 0.1f, 0.1f, 1.0f);

	//particleMoveShader.loadShader(GL_COMPUTE_SHADER, "../shaders/particle.comp");
	//particleMoveShader.compile();

	particleRenderShader.loadShader(GL_VERTEX_SHADER, "../shaders/particle.vert");
	particleRenderShader.loadShader(GL_FRAGMENT_SHADER, "../shaders/particle.frag");
	particleRenderShader.compile();

	// Initialize the particle position buffer
	glGenBuffers(1, &positionBuffer);
	glBindBuffer(GL_SHADER_STORAGE_BUFFER, positionBuffer);
	glBufferData(GL_SHADER_STORAGE_BUFFER, particlesNum * sizeof(xyzw), NULL, GL_DYNAMIC_DRAW);
	xyzw* pos = (xyzw*)glMapBufferRange(GL_SHADER_STORAGE_BUFFER, 0, particlesNum * sizeof(xyzw), GL_MAP_WRITE_BIT | GL_MAP_INVALIDATE_BUFFER_BIT);
	for (unsigned int i = 0; i < particlesNum; ++i)
	{
		pos[i].x = 2.0f * ((float)rand() / (float)RAND_MAX) - 1.0f;
		pos[i].y = 2.0f * ((float)rand() / (float)RAND_MAX) - 1.0f;
		pos[i].z = 2.0f * ((float)rand() / (float)RAND_MAX) - 1.0f;
		pos[i].w = 1.0f;
	}
	glUnmapBuffer(GL_SHADER_STORAGE_BUFFER);

	// Initialize the particle velocity buffer
	glGenBuffers(1, &velocityBuffer);
	glBindBuffer(GL_SHADER_STORAGE_BUFFER, velocityBuffer);
	glBufferData(GL_SHADER_STORAGE_BUFFER, particlesNum * sizeof(xyzw), NULL, GL_DYNAMIC_DRAW);
	xyzw* vel = (xyzw*)glMapBufferRange(GL_SHADER_STORAGE_BUFFER, 0, particlesNum * sizeof(xyzw), GL_MAP_WRITE_BIT | GL_MAP_INVALIDATE_BUFFER_BIT);
	for (unsigned int i = 0; i < particlesNum; ++i)
	{
		vel[i].x = 2.0f * ((float)rand() / (float)RAND_MAX) - 1.0f;
		vel[i].y = 2.0f * ((float)rand() / (float)RAND_MAX) - 1.0f;
		vel[i].z = 2.0f * ((float)rand() / (float)RAND_MAX) - 1.0f;
		vel[i].w = 0.0;
	}
	glUnmapBuffer(GL_SHADER_STORAGE_BUFFER);
	glBindBuffer(GL_SHADER_STORAGE_BUFFER, 0);

	// Initialize the vertex array object with the position and velocity buffers
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);

	glEnableVertexAttribArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, positionBuffer);
	glVertexAttribPointer((GLuint)0, 4, GL_FLOAT, GL_FALSE, sizeof(xyzw), (GLvoid*)0);

	glEnableVertexAttribArray(1);
	glBindBuffer(GL_ARRAY_BUFFER, velocityBuffer);
	glVertexAttribPointer((GLuint)1, 4, GL_FLOAT, GL_FALSE, sizeof(xyzw), (GLvoid*)0);

	glBindVertexArray(0);

	// Set point primitive size
	glPointSize(4.0f);


}

void onDisplay()
{

	void* dev_p = 0;
	void* dev_v = 0;

	size_t psize;
	size_t vsize;

	hipGraphicsGLRegisterBuffer(&p_res, positionBuffer, hipGraphicsRegisterFlagsNone);
	hipGraphicsGLRegisterBuffer(&v_res, velocityBuffer, hipGraphicsRegisterFlagsNone);

	hipGraphicsMapResources(1, &p_res);
	hipGraphicsMapResources(1, &v_res);

	hipGraphicsResourceGetMappedPointer(&dev_p, &psize, p_res);
	hipGraphicsResourceGetMappedPointer(&dev_v, &vsize, v_res);

	moveKernel << < particlesNum / 256, 256 >> > ((float*)dev_p, (float*)dev_v, particlesNum);

	float* lol = new float[particlesNum * 4];
	hipMemcpy(lol, dev_v, sizeof(float) * particlesNum * 4, hipMemcpyDeviceToHost);

	//for (int i = 0; i < particlesNum * 4; i++)
	//	printf("%f\n", lol[i]);

	delete[] lol;


	hipGraphicsUnmapResources(1, &p_res);
	hipGraphicsUnmapResources(1, &v_res);

	hipDeviceSynchronize();
	glFlush();

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	// Update position and velocity

	//Create CUDA variables
	glBindVertexArray(vao);
	// Render the particles
	particleRenderShader.enable();
	glDrawArrays(GL_POINTS, 0, particlesNum);
	glBindVertexArray(0);
	particleRenderShader.disable();

	glutSwapBuffers();
}

void onKeyboard(unsigned char key, int pX, int pY) {
	switch (key)
	{
	case 27:
		glutExit();
		break;
	}
}

void onDrag(int x, int y) {
	posX = (float(x) / glutGet(GLUT_WINDOW_WIDTH)) * 2.0f - 1.0f;
	posY = -((float(y) / glutGet(GLUT_WINDOW_HEIGHT)) * 2.0f - 1.0f);
}

void onIdle()
{
	glutPostRedisplay();
}

int main(int argc, char* argv[])
{
	glutInit(&argc, argv);

	glutInitContextVersion(4, 3);
	glutInitWindowSize(windowWidth, windowHeight);
	glutInitWindowPosition(100, 100);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
	glutCreateWindow(argv[0]);
	glewExperimental = true;
	glewInit();

	printf("GL Vendor    : %s\n", glGetString(GL_VENDOR));
	printf("GL Renderer  : %s\n", glGetString(GL_RENDERER));
	printf("GL Version (string)  : %s\n", glGetString(GL_VERSION));
	GLint major, minor;
	glGetIntegerv(GL_MAJOR_VERSION, &major);
	glGetIntegerv(GL_MINOR_VERSION, &minor);
	printf("GL Version (integer) : %d.%d\n", major, minor);
	printf("GLSL Version : %s\n", glGetString(GL_SHADING_LANGUAGE_VERSION));

	onInitialization();
	glutDisplayFunc(onDisplay);
	glutKeyboardFunc(onKeyboard);
	//glutMotionFunc(onDrag);
	//glutPassiveMotionFunc(onDrag);
	glutIdleFunc(onIdle);
	glutMainLoop();

	return 0;
}

